#include "hip/hip_runtime.h"
/**
@section LICENSE
Copyright (c) 2013-2016, Regents of the University of California
All rights reserved.

Redistribution and use in source and binary forms, with or without modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice, this list of conditions and the following disclaimer in the documentation and/or other materials provided with the distribution.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <stdio.h>
#include "kernel.h"
#include "pmcl3d_cons.h"

__constant__ float d_c1;
__constant__ float d_c2;
__constant__ float d_dth;
__constant__ float d_dt1;
__constant__ float d_dh1;
__constant__ float d_DT;
__constant__ float d_DH;
__constant__ int   d_nxt;
__constant__ int   d_nyt;
__constant__ int   d_nzt;
__constant__ int   d_slice_1;
__constant__ int   d_slice_2;
__constant__ int   d_yline_1;
__constant__ int   d_yline_2;

/* Strategy:
 * - Have file the converts CUDA to HIP (cuda_to_hip.h)
 * - Ensure that gets applied
 * - Compile the code
 * - Compiler flags might need to be changed, errors that need to be addressed
 * - Once it is compiling, need to make sure it works as expected
 * - Run an actual simulation to ensure that we get the same output within the floating point precision of error
 * - Output formats might be different; might need to compare by myself
      - Cybershake was built on this open source version
      - Our output is only velocity
      - Cybershake is SGT output
 * - Outputs are different from the two codes, be careful
*/

// Key difference between NVIDIA and HIP
// NVIDIA has a different number of threads (warp) than AMD
// NVIDIA warp size is 32, AMD warp size is 64
// This value isn't hardcoded, but if it had been it could be an issue with correctness or memory access
// Typically has performance benefits, want to pick number of threads_per_block to be a multiple of warp size

// There was a macro called "align" that is a macro to 32, but it was another keyword used in the HIP runtime library
// Some C++ include function had defined align, causing a conflict, safer to rename to awp_align.


// This couldn't be ported to HIP, instead of putting in texture memory, load it directly from global
texture<float, 1, hipReadModeElementType> p_vx1;
texture<float, 1, hipReadModeElementType> p_vx2;


void SetDeviceConstValue(float DH, float DT, int nxt, int nyt, int nzt)
{
    float h_c1, h_c2, h_dth, h_dt1, h_dh1;
    int   slice_1,  slice_2,  yline_1,  yline_2;
    h_c1  = 9.0/8.0;
    h_c2  = -1.0/24.0;
    h_dth = DT/DH;
    h_dt1 = 1.0/DT;
    h_dh1 = 1.0/DH;
    slice_1  = (nyt+4+8*loop)*(nzt+2*awp_align);
    slice_2  = (nyt+4+8*loop)*(nzt+2*awp_align)*2;
    yline_1  = nzt+2*awp_align;
    yline_2  = (nzt+2*awp_align)*2;

    hipMemcpyToSymbol(HIP_SYMBOL(d_c1),      &h_c1,    sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_c2),      &h_c2,    sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_dth),     &h_dth,   sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_dt1),     &h_dt1,   sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_dh1),     &h_dh1,   sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_DT),      &DT,      sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_DH),      &DH,      sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_nxt),     &nxt,     sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_nyt),     &nyt,     sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_nzt),     &nzt,     sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_slice_1), &slice_1, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_slice_2), &slice_2, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_yline_1), &yline_1, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_yline_2), &yline_2, sizeof(int));
    return;
}


void BindArrayToTexture(float* vx1, float* vx2, int memsize)
{
   hipBindTexture(0, p_vx1,  vx1,  memsize);
   hipBindTexture(0, p_vx2,  vx2,  memsize);
   hipDeviceSynchronize ();
   return;
}


void UnBindArrayFromTexture()
{
   hipUnbindTexture(p_vx1);
   hipUnbindTexture(p_vx2);
   return;
}


void dvelcx_H(float* u1,    float* v1,    float* w1,    float* xx,  float* yy, float* zz, float* xy,      float* xz, float* yz,
             float* dcrjx, float* dcrjy, float* dcrjz, float* d_1, int nyt,   int nzt,   hipStream_t St, int s_i,   int e_i)
{
    dim3 block (BLOCK_SIZE_Z, BLOCK_SIZE_Y, 1);
    dim3 grid ((nzt+BLOCK_SIZE_Z-1)/BLOCK_SIZE_Z, (nyt+BLOCK_SIZE_Y-1)/BLOCK_SIZE_Y,1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(dvelcx), hipFuncCachePreferL1);
    dvelcx<<<grid, block, 0, St>>>(u1, v1, w1, xx, yy, zz, xy, xz, yz, dcrjx, dcrjy, dcrjz, d_1, s_i, e_i);
    return;
}


void dvelcy_H(float* u1,       float* v1,    float* w1,    float* xx,  float* yy, float* zz, float* xy,   float* xz,   float* yz,
              float* dcrjx,    float* dcrjy, float* dcrjz, float* d_1, int nxt,   int nzt,   float* s_u1, float* s_v1, float* s_w1,
              hipStream_t St, int s_j,      int e_j,      int rank)
{
    if(rank==-1) return;
    dim3 block (BLOCK_SIZE_Z, BLOCK_SIZE_Y, 1);
    dim3 grid ((nzt+BLOCK_SIZE_Z-1)/BLOCK_SIZE_Z, (nxt+BLOCK_SIZE_Y-1)/BLOCK_SIZE_Y,1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(dvelcy), hipFuncCachePreferL1);
    dvelcy<<<grid, block, 0, St>>>(u1, v1, w1, xx, yy, zz, xy, xz, yz, dcrjx, dcrjy, dcrjz, d_1, s_u1, s_v1, s_w1, s_j, e_j);
    return;
}


void update_bound_y_H(float* u1,   float* v1, float* w1, float* f_u1,      float* f_v1,      float* f_w1,  float* b_u1, float* b_v1,
                      float* b_w1, int nxt,   int nzt,   hipStream_t St1, hipStream_t St2, int rank_f,  int rank_b)
{
     if(rank_f==-1 && rank_b==-1) return;
     dim3 block (BLOCK_SIZE_Z, BLOCK_SIZE_Y, 1);
     dim3 grid ((nzt+BLOCK_SIZE_Z-1)/BLOCK_SIZE_Z, (nxt+BLOCK_SIZE_Y-1)/BLOCK_SIZE_Y,1);
     hipFuncSetCacheConfig(reinterpret_cast<const void*>(update_boundary_y), hipFuncCachePreferL1);
     update_boundary_y<<<grid, block, 0, St1>>>(u1, v1, w1, f_u1, f_v1, f_w1, rank_f, Front);
     update_boundary_y<<<grid, block, 0, St2>>>(u1, v1, w1, b_u1, b_v1, b_w1, rank_b, Back);
     return;
}

// Possible argument misalignment, we are excluding one argument that doesn't seem to fit in our code (int rank)
void dstrqc_H(float* xx,       float* yy,     float* zz,    float* xy,    float* xz, float* yz,
              float* r1,       float* r2,     float* r3,    float* r4,    float* r5, float* r6,
              float* u1,       float* v1,     float* w1,    float* lam,   float* mu, float* qp,
              float* qs,       float* dcrjx,  float* dcrjy, float* dcrjz, int nyt,   int nzt,
              hipStream_t St, float* lam_mu, int NX,       int rankx,    int ranky, int  s_i,
              int e_i,         int s_j,       int e_j,      float* p_vx1, float* p_vx2)
{
    dim3 block (BLOCK_SIZE_Z, BLOCK_SIZE_Y, 1);
    dim3 grid ((nzt+BLOCK_SIZE_Z-1)/BLOCK_SIZE_Z, (e_j-s_j+1+BLOCK_SIZE_Y-1)/BLOCK_SIZE_Y,1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(dstrqc), hipFuncCachePreferL1);
    dstrqc<<<grid, block, 0, St>>>(xx,    yy,    zz,  xy,  xz, yz, r1, r2,    r3,    r4,    r5,     r6,
                                   u1,    v1,    w1,  lam, mu, qp, qs, dcrjx, dcrjy, dcrjz, lam_mu, NX,
                                   rankx, ranky, s_i, e_i, s_j, e_j, p_vx1, p_vx2);
    return;
}


void addsrc_H(int i,      int READ_STEP, int dim,    int* psrc,  int npsrc,  hipStream_t St,
              float* axx, float* ayy,    float* azz, float* axz, float* ayz, float* axy,
              float* xx,  float* yy,     float* zz,  float* xy,  float* yz,  float* xz)
{
    dim3 grid, block;
    if(npsrc < 256)
    {
       block.x = npsrc;
       grid.x = 1;
    }
    else
    {
       block.x = 256;
       grid.x  = int((npsrc+255)/256);
    }
    hipError_t cerr;
    cerr=hipGetLastError();
    if(cerr!=hipSuccess) printf("CUDA ERROR: addsrc before kernel: %s\n",hipGetErrorString(cerr));
    addsrc_cu<<<grid, block, 0, St>>>(i,  READ_STEP, dim, psrc, npsrc, axx, ayy, azz, axz, ayz, axy,
                                      xx, yy,        zz,  xy,   yz,  xz);
    cerr=hipGetLastError();
    if(cerr!=hipSuccess) printf("CUDA ERROR: addsrc after kernel: %s\n",hipGetErrorString(cerr));
    return;
}


__global__ void dvelcx(float* u1,    float* v1,    float* w1,    float* xx, float* yy, float* zz, float* xy, float* xz, float* yz,
                      float* dcrjx, float* dcrjy, float* dcrjz, float* d_1, int s_i,   int e_i)
{
    register int   i, j, k, pos,     pos_im1, pos_im2;
    register int   pos_km2, pos_km1, pos_kp1, pos_kp2;
    register int   pos_jm2, pos_jm1, pos_jp1, pos_jp2;
    register int   pos_ip1, pos_jk1, pos_ik1, pos_ijk;
    register float f_xx,    xx_im1,  xx_ip1,  xx_im2;
    register float f_xy,    xy_ip1,  xy_ip2,  xy_im1;
    register float f_xz,    xz_ip1,  xz_ip2,  xz_im1;
    register float f_d1,    f_d2,    f_d3,    f_dcrj, f_dcrjy, f_dcrjz, f_yz;

    k    = blockIdx.x*BLOCK_SIZE_Z+threadIdx.x+awp_align;
    j    = blockIdx.y*BLOCK_SIZE_Y+threadIdx.y+2+4*loop;
    i    = e_i;
    pos  = i*d_slice_1+j*d_yline_1+k;

    f_xx    = xx[pos+d_slice_1];
    xx_im1  = xx[pos];
    xx_im2  = xx[pos-d_slice_1];
    xy_ip1  = xy[pos+d_slice_2];
    f_xy    = xy[pos+d_slice_1];
    xy_im1  = xy[pos];
    xz_ip1  = xz[pos+d_slice_2];
    f_xz    = xz[pos+d_slice_1];
    xz_im1  = xz[pos];
    f_dcrjz = dcrjz[k];
    f_dcrjy = dcrjy[j];
    for(i=e_i;i>=s_i;i--)
    {
        pos_km2  = pos-2;
        pos_km1  = pos-1;
        pos_kp1  = pos+1;
        pos_kp2  = pos+2;
        pos_jm2  = pos-d_yline_2;
        pos_jm1  = pos-d_yline_1;
        pos_jp1  = pos+d_yline_1;
        pos_jp2  = pos+d_yline_2;
        pos_im1  = pos-d_slice_1;
        pos_im2  = pos-d_slice_2;
        pos_ip1  = pos+d_slice_1;
        pos_jk1  = pos-d_yline_1-1;
        pos_ik1  = pos+d_slice_1-1;
        pos_ijk  = pos+d_slice_1-d_yline_1;

        xx_ip1   = f_xx;
        f_xx     = xx_im1;
        xx_im1   = xx_im2;
        xx_im2   = xx[pos_im2];
        xy_ip2   = xy_ip1;
        xy_ip1   = f_xy;
        f_xy     = xy_im1;
        xy_im1   = xy[pos_im1];
        xz_ip2   = xz_ip1;
        xz_ip1   = f_xz;
        f_xz     = xz_im1;
        xz_im1   = xz[pos_im1];
        f_yz     = yz[pos];

        f_dcrj   = dcrjx[i]*f_dcrjy*f_dcrjz;
        f_d1     = 0.25*(d_1[pos] + d_1[pos_jm1] + d_1[pos_km1] + d_1[pos_jk1]);
        f_d2     = 0.25*(d_1[pos] + d_1[pos_ip1] + d_1[pos_km1] + d_1[pos_ik1]);
        f_d3     = 0.25*(d_1[pos] + d_1[pos_ip1] + d_1[pos_jm1] + d_1[pos_ijk]);

        f_d1     = d_dth/f_d1;
        f_d2     = d_dth/f_d2;
	f_d3     = d_dth/f_d3;

    	u1[pos]  = (u1[pos] + f_d1*( d_c1*(f_xx        - xx_im1)      + d_c2*(xx_ip1      - xx_im2)
                                   + d_c1*(f_xy        - xy[pos_jm1]) + d_c2*(xy[pos_jp1] - xy[pos_jm2])
                                   + d_c1*(f_xz        - xz[pos_km1]) + d_c2*(xz[pos_kp1] - xz[pos_km2]) ))*f_dcrj;
        v1[pos]  = (v1[pos] + f_d2*( d_c1*(xy_ip1      - f_xy)        + d_c2*(xy_ip2      - xy_im1)
                                   + d_c1*(yy[pos_jp1] - yy[pos])     + d_c2*(yy[pos_jp2] - yy[pos_jm1])
                                   + d_c1*(f_yz        - yz[pos_km1]) + d_c2*(yz[pos_kp1] - yz[pos_km2]) ))*f_dcrj;

        w1[pos]  = (w1[pos] + f_d3*( d_c1*(xz_ip1      - f_xz)        + d_c2*(xz_ip2      - xz_im1)
                                   + d_c1*(f_yz        - yz[pos_jm1]) + d_c2*(yz[pos_jp1] - yz[pos_jm2])
                                   + d_c1*(zz[pos_kp1] - zz[pos])     + d_c2*(zz[pos_kp2] - zz[pos_km1]) ))*f_dcrj;
        pos      = pos_im1;
    }

    return;
}


__global__ void dvelcy(float* u1,    float* v1,    float* w1,    float* xx,  float* yy,   float* zz,   float* xy, float* xz, float* yz,
                       float* dcrjx, float* dcrjy, float* dcrjz, float* d_1, float* s_u1, float* s_v1, float* s_w1, int s_j,   int e_j)
{
    register int   i, j, k, pos,     j2,      pos2, pos_jm1, pos_jm2;
    register int   pos_km2, pos_km1, pos_kp1, pos_kp2;
    register int   pos_im2, pos_im1, pos_ip1, pos_ip2;
    register int   pos_jk1, pos_ik1, pos_ijk;
    register float f_xy,    xy_jp1,  xy_jm1,  xy_jm2;
    register float f_yy,    yy_jp2,  yy_jp1,  yy_jm1;
    register float f_yz,    yz_jp1,  yz_jm1,  yz_jm2;
    register float f_d1,    f_d2,    f_d3,    f_dcrj, f_dcrjx, f_dcrjz, f_xz;

    k     = blockIdx.x*BLOCK_SIZE_Z+threadIdx.x+awp_align;
    i     = blockIdx.y*BLOCK_SIZE_Y+threadIdx.y+2+4*loop;
    j     = e_j;
    j2    = 4*loop-1;
    pos   = i*d_slice_1+j*d_yline_1+k;
    pos2  = i*4*loop*d_yline_1+j2*d_yline_1+k;

    f_xy    = xy[pos+d_yline_1];
    xy_jm1  = xy[pos];
    xy_jm2  = xy[pos-d_yline_1];
    yy_jp1  = yy[pos+d_yline_2];
    f_yy    = yy[pos+d_yline_1];
    yy_jm1  = yy[pos];
    f_yz    = yz[pos+d_yline_1];
    yz_jm1  = yz[pos];
    yz_jm2  = yz[pos-d_yline_1];
    f_dcrjz = dcrjz[k];
    f_dcrjx = dcrjx[i];
    for(j=e_j; j>=s_j; j--)
    {
        pos_km2  = pos-2;
        pos_km1  = pos-1;
        pos_kp1  = pos+1;
        pos_kp2  = pos+2;
        pos_jm2  = pos-d_yline_2;
        pos_jm1  = pos-d_yline_1;
        pos_im1  = pos-d_slice_1;
        pos_im2  = pos-d_slice_2;
        pos_ip1  = pos+d_slice_1;
        pos_ip2  = pos+d_slice_2;
        pos_jk1  = pos-d_yline_1-1;
        pos_ik1  = pos+d_slice_1-1;
        pos_ijk  = pos+d_slice_1-d_yline_1;

        xy_jp1   = f_xy;
        f_xy     = xy_jm1;
        xy_jm1   = xy_jm2;
        xy_jm2   = xy[pos_jm2];
        yy_jp2   = yy_jp1;
        yy_jp1   = f_yy;
        f_yy     = yy_jm1;
        yy_jm1   = yy[pos_jm1];
        yz_jp1   = f_yz;
        f_yz     = yz_jm1;
        yz_jm1   = yz_jm2;
        yz_jm2   = yz[pos_jm2];
        f_xz     = xz[pos];

        f_dcrj   = f_dcrjx*dcrjy[j]*f_dcrjz;
        f_d1     = 0.25*(d_1[pos] + d_1[pos_jm1] + d_1[pos_km1] + d_1[pos_jk1]);
        f_d2     = 0.25*(d_1[pos] + d_1[pos_ip1] + d_1[pos_km1] + d_1[pos_ik1]);
        f_d3     = 0.25*(d_1[pos] + d_1[pos_ip1] + d_1[pos_jm1] + d_1[pos_ijk]);

        f_d1     = d_dth/f_d1;
        f_d2     = d_dth/f_d2;
        f_d3     = d_dth/f_d3;

        s_u1[pos2] = (u1[pos] + f_d1*( d_c1*(xx[pos]     - xx[pos_im1]) + d_c2*(xx[pos_ip1] - xx[pos_im2])
                                     + d_c1*(f_xy        - xy_jm1)      + d_c2*(xy_jp1      - xy_jm2)
                                     + d_c1*(f_xz        - xz[pos_km1]) + d_c2*(xz[pos_kp1] - xz[pos_km2]) ))*f_dcrj;
        s_v1[pos2] = (v1[pos] + f_d2*( d_c1*(xy[pos_ip1] - f_xy)        + d_c2*(xy[pos_ip2] - xy[pos_im1])
                                     + d_c1*(yy_jp1      - f_yy)        + d_c2*(yy_jp2      - yy_jm1)
                                     + d_c1*(f_yz        - yz[pos_km1]) + d_c2*(yz[pos_kp1] - yz[pos_km2]) ))*f_dcrj;
        s_w1[pos2] = (w1[pos] + f_d3*( d_c1*(xz[pos_ip1] - f_xz)        + d_c2*(xz[pos_ip2] - xz[pos_im1])
                                     + d_c1*(f_yz        - yz_jm1)      + d_c2*(yz_jp1      - yz_jm2)
                                     + d_c1*(zz[pos_kp1] - zz[pos])     + d_c2*(zz[pos_kp2] - zz[pos_km1]) ))*f_dcrj;

        pos        = pos_jm1;
        pos2       = pos2 - d_yline_1;
    }
    return;
}

__global__ void update_boundary_y(float* u1, float* v1, float* w1, float* s_u1, float* s_v1, float* s_w1, int rank, int flag)
{
    register int i, j, k, pos, posj;
    k     = blockIdx.x*BLOCK_SIZE_Z+threadIdx.x+awp_align;
    i     = blockIdx.y*BLOCK_SIZE_Y+threadIdx.y+2+4*loop;

    if(flag==Front && rank!=-1){
	j     = 2;
    	pos   = i*d_slice_1+j*d_yline_1+k;
        posj  = i*4*loop*d_yline_1+k;
	for(j=2;j<2+4*loop;j++){
		u1[pos] = s_u1[posj];
		v1[pos] = s_v1[posj];
		w1[pos] = s_w1[posj];
		pos	= pos  + d_yline_1;
  		posj	= posj + d_yline_1;
	}
    }

    if(flag==Back && rank!=-1){
    	j     = d_nyt+4*loop+2;
    	pos   = i*d_slice_1+j*d_yline_1+k;
        posj  = i*4*loop*d_yline_1+k;
	for(j=d_nyt+4*loop+2;j<d_nyt+8*loop+2;j++){
	        u1[pos] = s_u1[posj];
                v1[pos] = s_v1[posj];
                w1[pos] = s_w1[posj];
                pos     = pos  + d_yline_1;
                posj    = posj + d_yline_1;
	}
    }
    return;
}

__global__ void dstrqc(float* xx, float* yy,    float* zz,    float* xy,    float* xz,     float* yz,
                       float* r1, float* r2,    float* r3,    float* r4,    float* r5,     float* r6,
                       float* u1, float* v1,    float* w1,    float* lam,   float* mu,     float* qp,
                       float* qs, float* dcrjx, float* dcrjy, float* dcrjz, float* lam_mu, int NX,
                       int rankx, int ranky,    int s_i,      int e_i,      int s_j,       float* p_vx1,
                       float* p_vx2)
{
    register int   i,  j,  k,  g_i;
    register int   pos,     pos_ip1, pos_im2, pos_im1;
    register int   pos_km2, pos_km1, pos_kp1, pos_kp2;
    register int   pos_jm2, pos_jm1, pos_jp1, pos_jp2;
    register int   pos_ik1, pos_jk1, pos_ijk, pos_ijk1;
    register float vs1, vs2, vs3, a1, tmp, vx1;
    register float xl,  xm,  xmu1, xmu2, xmu3;
    register float qpa, h,   h1,   h2,   h3;
    register float f_vx1, f_vx2,  f_dcrj, f_r,  f_dcrjy, f_dcrjz;
    register float f_u1, u1_ip1, u1_ip2, u1_im1;
    register float f_v1, v1_im1, v1_ip1, v1_im2;
    register float f_w1, w1_im1, w1_im2, w1_ip1;

    k    = blockIdx.x*BLOCK_SIZE_Z+threadIdx.x+awp_align;
    j    = blockIdx.y*BLOCK_SIZE_Y+threadIdx.y+s_j;
    i    = e_i;
    pos  = i*d_slice_1+j*d_yline_1+k;

    u1_ip1 = u1[pos+d_slice_2];
    f_u1   = u1[pos+d_slice_1];
    u1_im1 = u1[pos];
    f_v1   = v1[pos+d_slice_1];
    v1_im1 = v1[pos];
    v1_im2 = v1[pos-d_slice_1];
    f_w1   = w1[pos+d_slice_1];
    w1_im1 = w1[pos];
    w1_im2 = w1[pos-d_slice_1];
    f_dcrjz = dcrjz[k];
    f_dcrjy = dcrjy[j];
    for(i=e_i;i>=s_i;i--)
    {
//        f_vx1    = tex1Dfetch(p_vx1, pos);
//        f_vx2    = tex1Dfetch(p_vx2, pos);
        f_vx1    = p_vx1[pos];
        f_vx2    = p_vx2[pos];
        f_dcrj   = dcrjx[i]*f_dcrjy*f_dcrjz;

        pos_km2  = pos-2;
        pos_km1  = pos-1;
        pos_kp1  = pos+1;
        pos_kp2  = pos+2;
        pos_jm2  = pos-d_yline_2;
        pos_jm1  = pos-d_yline_1;
        pos_jp1  = pos+d_yline_1;
        pos_jp2  = pos+d_yline_2;
        pos_im2  = pos-d_slice_2;
        pos_im1  = pos-d_slice_1;
        pos_ip1  = pos+d_slice_1;
        pos_jk1  = pos-d_yline_1-1;
        pos_ik1  = pos+d_slice_1-1;
        pos_ijk  = pos+d_slice_1-d_yline_1;
        pos_ijk1 = pos+d_slice_1-d_yline_1-1;

        xl       = 8.0/(  lam[pos]      + lam[pos_ip1] + lam[pos_jm1] + lam[pos_ijk]
                        + lam[pos_km1]  + lam[pos_ik1] + lam[pos_jk1] + lam[pos_ijk1] );
        xm       = 16.0/( mu[pos]       + mu[pos_ip1]  + mu[pos_jm1]  + mu[pos_ijk]
                        + mu[pos_km1]   + mu[pos_ik1]  + mu[pos_jk1]  + mu[pos_ijk1] );
        xmu1     = 2.0/(  mu[pos]       + mu[pos_km1] );
        xmu2     = 2.0/(  mu[pos]       + mu[pos_jm1] );
        xmu3     = 2.0/(  mu[pos]       + mu[pos_ip1] );
        xl       = xl  +  xm;
        qpa      = 0.0625*( qp[pos]     + qp[pos_ip1] + qp[pos_jm1] + qp[pos_ijk]
                          + qp[pos_km1] + qp[pos_ik1] + qp[pos_jk1] + qp[pos_ijk1] );
        h        = 0.0625*( qs[pos]     + qs[pos_ip1] + qs[pos_jm1] + qs[pos_ijk]
                          + qs[pos_km1] + qs[pos_ik1] + qs[pos_jk1] + qs[pos_ijk1] );
        h1       = 0.250*(  qs[pos]     + qs[pos_km1] );
        h2       = 0.250*(  qs[pos]     + qs[pos_jm1] );
        h3       = 0.250*(  qs[pos]     + qs[pos_ip1] );

        h        = -xm*h*d_dh1;
        h1       = -xmu1*h1*d_dh1;
        h2       = -xmu2*h2*d_dh1;
        h3       = -xmu3*h3*d_dh1;
        qpa      = -qpa*xl*d_dh1;
        xm       = xm*d_dth;
        xmu1     = xmu1*d_dth;
        xmu2     = xmu2*d_dth;
        xmu3     = xmu3*d_dth;
        xl       = xl*d_dth;
        f_vx2    = f_vx2*f_vx1;
        h        = h*f_vx1;
        h1       = h1*f_vx1;
        h2       = h2*f_vx1;
        h3       = h3*f_vx1;
        qpa      = qpa*f_vx1;

        xm       = xm+d_DT*h;
        xmu1     = xmu1+d_DT*h1;
        xmu2     = xmu2+d_DT*h2;
        xmu3     = xmu3+d_DT*h3;
        vx1      = d_DT*(1+f_vx2);

        u1_ip2   = u1_ip1;
        u1_ip1   = f_u1;
        f_u1     = u1_im1;
        u1_im1   = u1[pos_im1];
        v1_ip1   = f_v1;
        f_v1     = v1_im1;
        v1_im1   = v1_im2;
        v1_im2   = v1[pos_im2];
        w1_ip1   = f_w1;
        f_w1     = w1_im1;
        w1_im1   = w1_im2;
        w1_im2   = w1[pos_im2];

        if(k == d_nzt+awp_align-1)
        {
		u1[pos_kp1] = f_u1 - (f_w1        - w1_im1);
    		v1[pos_kp1] = f_v1 - (w1[pos_jp1] - f_w1);

                g_i  = d_nxt*rankx + i - 4*loop - 1;

    		if(g_i<NX)
        		vs1	= u1_ip1 - (w1_ip1    - f_w1);
    		else
        		vs1	= 0.0;

                g_i  = d_nyt*ranky + j - 4*loop - 1;
    		if(g_i>1)
        		vs2	= v1[pos_jm1] - (f_w1 - w1[pos_jm1]);
    		else
        		vs2	= 0.0;

    		w1[pos_kp1]	= w1[pos_km1] - lam_mu[i*(d_nyt+4+8*loop) + j]*((vs1         - u1[pos_kp1]) + (u1_ip1 - f_u1)
                                      +     			                (v1[pos_kp1] - vs2)         + (f_v1   - v1[pos_jm1]) );
        }
	else if(k == d_nzt+awp_align-2)
	{
                u1[pos_kp2] = u1[pos_kp1] - (w1[pos_kp1]   - w1[pos_im1+1]);
                v1[pos_kp2] = v1[pos_kp1] - (w1[pos_jp1+1] - w1[pos_kp1]);
	}

    	vs1      = d_c1*(u1_ip1 - f_u1)        + d_c2*(u1_ip2      - u1_im1);
        vs2      = d_c1*(f_v1   - v1[pos_jm1]) + d_c2*(v1[pos_jp1] - v1[pos_jm2]);
        vs3      = d_c1*(f_w1   - w1[pos_km1]) + d_c2*(w1[pos_kp1] - w1[pos_km2]);

        tmp      = xl*(vs1+vs2+vs3);
        a1       = qpa*(vs1+vs2+vs3);
        tmp      = tmp+d_DT*a1;

        f_r      = r1[pos];
        xx[pos]  = (xx[pos]  + tmp - xm*(vs2+vs3) + vx1*f_r)*f_dcrj;
        r1[pos]  = f_vx2*f_r - h*(vs2+vs3)        + a1;
        f_r      = r2[pos];
        yy[pos]  = (yy[pos]  + tmp - xm*(vs1+vs3) + vx1*f_r)*f_dcrj;
        r2[pos]  = f_vx2*f_r - h*(vs1+vs3)        + a1;
        f_r      = r3[pos];
        zz[pos]  = (zz[pos]  + tmp - xm*(vs1+vs2) + vx1*f_r)*f_dcrj;
        r3[pos]  = f_vx2*f_r - h*(vs1+vs2)        + a1;

        vs1      = d_c1*(u1[pos_jp1] - f_u1)   + d_c2*(u1[pos_jp2] - u1[pos_jm1]);
        vs2      = d_c1*(f_v1        - v1_im1) + d_c2*(v1_ip1      - v1_im2);
        f_r      = r4[pos];
        xy[pos]  = (xy[pos]  + xmu1*(vs1+vs2) + vx1*f_r)*f_dcrj;
        r4[pos]  = f_vx2*f_r + h1*(vs1+vs2);

        if(k == d_nzt+awp_align-1)
        {
                zz[pos+1] = -zz[pos];
        	xz[pos]   = 0.0;
                yz[pos]   = 0.0;
        }
        else
        {
        	vs1     = d_c1*(u1[pos_kp1] - f_u1)   + d_c2*(u1[pos_kp2] - u1[pos_km1]);
        	vs2     = d_c1*(f_w1        - w1_im1) + d_c2*(w1_ip1      - w1_im2);
        	f_r     = r5[pos];
        	xz[pos] = (xz[pos]  + xmu2*(vs1+vs2) + vx1*f_r)*f_dcrj;
        	r5[pos] = f_vx2*f_r + h2*(vs1+vs2);


        	vs1     = d_c1*(v1[pos_kp1] - f_v1) + d_c2*(v1[pos_kp2] - v1[pos_km1]);
        	vs2     = d_c1*(w1[pos_jp1] - f_w1) + d_c2*(w1[pos_jp2] - w1[pos_jm1]);
        	f_r     = r6[pos];
        	yz[pos] = (yz[pos]  + xmu3*(vs1+vs2) + vx1*f_r)*f_dcrj;
        	r6[pos] = f_vx2*f_r + h3*(vs1+vs2);

                if(k == d_nzt+awp_align-2)
                {
                    zz[pos+3] = -zz[pos];
                    xz[pos+2] = -xz[pos];
                    yz[pos+2] = -yz[pos];
		}
		else if(k == d_nzt+awp_align-3)
		{
                    xz[pos+4] = -xz[pos];
                    yz[pos+4] = -yz[pos];
		}
 	}
        pos     = pos_im1;
    }
    return;
}


__global__ void addsrc_cu(int i,      int READ_STEP, int dim,    int* psrc,  int npsrc,
                          float* axx, float* ayy,    float* azz, float* axz, float* ayz, float* axy,
                          float* xx,  float* yy,     float* zz,  float* xy,  float* yz,  float* xz)
{
        register float vtst;
        register int idx, idy, idz, j, pos;
        j = blockIdx.x*blockDim.x+threadIdx.x;
        if(j >= npsrc) return;
        vtst = (float)d_DT/(d_DH*d_DH*d_DH);

        i   = i - 1;
        idx = psrc[j*dim]   + 1 + 4*loop;
        idy = psrc[j*dim+1] + 1 + 4*loop;
        idz = psrc[j*dim+2] + awp_align - 1;
        pos = idx*d_slice_1 + idy*d_yline_1 + idz;

        xx[pos] = xx[pos] - vtst*axx[j*READ_STEP+i];
        yy[pos] = yy[pos] - vtst*ayy[j*READ_STEP+i];
        zz[pos] = zz[pos] - vtst*azz[j*READ_STEP+i];
        xz[pos] = xz[pos] - vtst*axz[j*READ_STEP+i];
        yz[pos] = yz[pos] - vtst*ayz[j*READ_STEP+i];
        xy[pos] = xy[pos] - vtst*axy[j*READ_STEP+i];

        return;
}
